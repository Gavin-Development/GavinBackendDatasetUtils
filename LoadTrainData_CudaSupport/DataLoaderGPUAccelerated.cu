#include "hip/hip_runtime.h"
#include "DataLoader.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>


__global__ void CudaKernel(char* FileData, int64_t* NumberOfLinesArr, int64_t* NumberOfLinesIO, size_t n) {
	int Tid = threadIdx.x;
	if (Tid < n) {
		if (FileData[Tid] == (char)0x0A) {
			NumberOfLinesArr[Tid] = 1;
		}
		else NumberOfLinesArr[Tid] = 0;

		for (int stride = 1; stride < n; stride * 2) {
			if (Tid + stride < n / stride) {
				NumberOfLinesArr[Tid] += NumberOfLinesArr[Tid + stride];
			}
		}

		if (Tid == 0) {
			*NumberOfLinesIO = NumberOfLinesArr[0];
		}
	}
};

void LaunchCudaKernels(char* FileData, size_t FileLength) {
	int64_t* NumberOfLines;
	int64_t* Device_NumberOfLines;
	char* Device_FileContentsBuffer;
	int64_t* Device_NumberOfLinesArr;
	hipMallocManaged(&Device_FileContentsBuffer, sizeof(char) * FileLength);
	hipMallocManaged(&Device_NumberOfLinesArr, sizeof(int64_t) * FileLength);
	hipMallocManaged(&Device_NumberOfLines, sizeof(int64_t));
	NumberOfLines = (int64_t*)malloc(sizeof(int64_t) * FileLength);

	hipMemcpy(Device_FileContentsBuffer, FileData, sizeof(char) * FileLength, hipMemcpyHostToDevice);

	CudaKernel <<<1, FileLength>>> (Device_FileContentsBuffer, Device_NumberOfLinesArr, Device_NumberOfLines, FileLength);
	hipDeviceSynchronize();

	hipMemcpy(NumberOfLines, Device_NumberOfLinesArr, sizeof(int64_t) * FileLength, hipMemcpyDeviceToHost);
	
	for (int64_t i = 0; i < FileLength; i++) {
		if (NumberOfLines[i] == 1) {
			std::cout << "NewLine." << std::endl;
		}
	}
	hipFree(Device_FileContentsBuffer);
	hipFree(Device_NumberOfLinesArr);
	hipFree(Device_NumberOfLines);
	std::cout << *NumberOfLines << std::endl;
	std::cout << FileLength << std::endl;
	char tmp[3] = { FileData[1], FileData[1 + 1], 0 };
	std::cout << tmp << std::endl;
	std::cout << "Cuda Kernels Complete." << std::endl;
};

std::vector<py::list> LoadTrainDataGPU_Accelerated_Future(int64_t samplesToRead, std::string dataPath, std::string tokenizerName, int startToken, int endToken, int sampleLength, int paddingValue) {
	std::cout << "WARNING THI FUNCTION MIGHT NOT WORK AS INTENDED DO NOT USE PROPERLY." << std::endl;
	std::vector<py::list> FileData;
	if (samplesToRead < 100) {
		std::cout << "Please Specify A MINIMUM Of 100 Samples To Load." << std::endl;
		return FileData;
	}
	int64_t StartTime = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
	int64_t EndTime;
	int64_t TimeTaken;
	std::string FileName = dataPath + tokenizerName;
	std::ifstream File;
	int64_t MaxSamples = samplesToRead;
	int64_t ProgressReportInterval = MaxSamples / 100;

	std::cout << "Loading " << samplesToRead << " Samples From " << FileName << std::endl;

	File = std::ifstream(FileName, std::ios::binary | std::ios::ate);
	if (File.is_open()) {
		std::cout << "Loading (Up To) " << MaxSamples << " Samples." << std::endl;
	}
	else {
		std::cout << "Failed To Open File." << std::endl;
		return FileData;
	}
	char* FileContentsBuffer;
	size_t FileLength = File.tellg();
	File.seekg(0);
	FileContentsBuffer = (char*)malloc(sizeof(char) * FileLength);
	File.read(FileContentsBuffer, FileLength);

	LaunchCudaKernels(FileContentsBuffer, FileLength);

	for (int64_t i = 0; i < 1000; i++) {
		const char tmp[2] = { FileContentsBuffer[i], 0 };
		std::cout << tmp;
		if (FileContentsBuffer[i] == (char)0x0A) {
			std::cout << "Found A New Line Char." << std::endl;
		}
	}
	std::cout << std::endl;

	for (int i = 0; i < 1000; i++) {
		std::cout << FileContentsBuffer[i];
	}
	std::cout << std::endl;

	File.close();
	std::cout << "Samples Have Been Read." << std::endl;
	EndTime = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
	TimeTaken = (EndTime - StartTime) / 1000000000;
	std::cout << "Time Taken: " << TimeTaken << " Seconds." << std::endl;
	return FileData;
}